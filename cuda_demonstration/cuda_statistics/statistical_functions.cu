#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>

hipError_t cudaMean1d(int* a, int *b, const int size);

__global__ void sum1dKernel(int *a, int *val) {
	int i = threadIdx.x;
	*val += a[i];
	printf("%d %d\n", a[i], *val);
}

int main() {
	const int m = 5;
	int arr[m] = { 1, 2, 3, 4, 5 };
	int res = -1;

	hipError_t status = cudaMean1d(arr, &res, m);
	printf("\n");

	if (status != hipSuccess) {
		fprintf(stderr, "Something went wrong\n");
		return 1;
	}

	printf("Mean value of a {1, 2, 3, 4, 5} array is %d\n", res);

	status = hipDeviceReset();
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to reset device\n");
		return 1;
	}

	return 0;
}

hipError_t cudaMean1d(int* a, int* b, const int size) {
	int *dev_a = 0;
	int *dev_c = 0;

	hipError_t status;

	status = hipSetDevice(0);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to start device. Do you have CUDA device installed?\n");
		goto Error;
	}

	status = hipMalloc((void**)&dev_a, size * sizeof(int));

	if (status != hipSuccess) {
		fprintf(stderr, "Failed to allocate memory\n");
		goto Error;
	}

	status = hipMalloc((void**)&dev_c, sizeof(int));

	if (status != hipSuccess) {
		fprintf(stderr, "Failed to allocate memory\n");
		goto Error;
	}

	status = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to copy from host to GPU buffer\n");
		goto Error;
	}

	sum1dKernel << <1, size >> > (dev_a, dev_c);

	status = hipGetLastError();
	if (status != hipSuccess) {
		fprintf(stderr, "Last error message %s\n", hipGetErrorString(status));
		goto Error;
	}

	status = hipDeviceSynchronize();
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to synchronize device. Error %d\n", status);
		goto Error;
	}

	status = hipMemcpy(b, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "Failed to copy from GPU buffer to host\n");
		goto Error;
	}

Error:
	hipFree(dev_a);
	return status;
}